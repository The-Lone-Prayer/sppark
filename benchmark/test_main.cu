#include "../util/gpu_t.cuh"


int main() {
    printf("%d\n", cuda_available());
    int count = 9999;
    hipGetDeviceCount(&count);
    printf("%d\n", count);

    auto prop = gpu_props(0);
    printf("%s\n", prop.name);
}
